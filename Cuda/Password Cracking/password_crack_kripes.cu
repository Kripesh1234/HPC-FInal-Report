
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o password_kripes password_crack_kripes.cu


     To Run:
     ./password_kripes > resultpwd_cuda_kripes.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "KD5698";
  char plain_password2[] = "AC1623";
  char plain_password3[] = "EF0126";
  char plain_password4[] = "KL2589";

  char *k = attempt;
  char *r = attempt;
  char *i = attempt;
  char *p = attempt;
  char *k1 = plain_password1;
  char *k2 = plain_password2;
  char *k3 = plain_password3;
  char *k4 = plain_password4;

  while(*k == *k1) {
   if(*k == '\0')
    {
    printf("Password: %s\n",plain_password1);
      break;
    }

    k++;
    k1++;
  }
    
  while(*r == *k2) {
   if(*r == '\0')
    {
    printf("Password: %s\n",plain_password2);
      break;
}

    r++;
    k2++;
  }

  while(*i == *k3) {
   if(*i == '\0')
    {
    printf("Password: %s\n",plain_password3);
      break;
    }

    i++;
    k3++;
  }

  while(*p == *k4) {
   if(*p == '\0')
    {
    printf("Password: %s\n",plain_password4);
      return 1;
    }

    p++;
    k4++;
  }
  return 0;

}
__global__ void  kernel() {
char e,f,g,h;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(e='0'; e<='9'; e++){
      for(f='0'; f<='9'; f++){
        for(g='0';g<='9';g++){
          for(h='0';h<='9';h++){
            password[2] = e;
            password[3] = f;
            password[4]= g;
            password[5]=h;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



